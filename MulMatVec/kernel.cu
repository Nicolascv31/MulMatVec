#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <hip/hip_runtime.h>

__global__ void matVecMul(const int* A, const int* x, int* y, int M, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M) {
        int sum = 0;
        for (int j = 0; j < N; ++j) {
            sum += A[row * N + j] * x[j];
        }
        y[row] = sum;
    }
}

int main() {
    int M = 10; 
    int N = 10; 

    size_t size_A = M * N * sizeof(int);
    size_t size_x = N * sizeof(int);
    size_t size_y = M * sizeof(int);

    
    int* h_A = (int*)malloc(size_A);
    int* h_x = (int*)malloc(size_x);
    int* h_y = (int*)malloc(size_y);

    
    for (int i = 0; i < M * N; ++i) h_A[i] = 6;
    for (int i = 0; i < N; ++i) h_x[i] = 2;

    
    int* d_A, * d_x, * d_y;
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_x, size_x);
    hipMalloc((void**)&d_y, size_y);

    
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, size_x, hipMemcpyHostToDevice);

   
    int threadsPerBlock = 256;
    int blocksPerGrid = (M + threadsPerBlock - 1) / threadsPerBlock;
    matVecMul << <blocksPerGrid, threadsPerBlock >> > (d_A, d_x, d_y, M, N);

    
    hipMemcpy(h_y, d_y, size_y, hipMemcpyDeviceToHost);

    
    std::cout << "Resultado del producto matriz * vector:" << std::endl;
    for (int i = 0; i < M; ++i) {
        std::cout << h_y[i] << " ";
    }
    std::cout << std::endl;

    
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    free(h_A);
    free(h_x);
    free(h_y);

    return 0;
}
